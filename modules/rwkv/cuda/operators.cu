#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "ATen/ATen.h"
#include <hip/hip_fp16.h>
#define MIN_VALUE (-1e38)
typedef at::Half fp16;

__global__ void kernel_wkv_forward(const int B, const int T, const int C,
                               const float *__restrict__ const _w, const float *__restrict__ const _u, const fp16 *__restrict__ const _k, const fp16 *__restrict__ const _v,
                               fp16 *__restrict__ const _y, float *__restrict__ const _aa, float *__restrict__ const _bb, float *__restrict__ const _pp) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;
    const int _state_offset = _b * C + _c;

    float u = _u[_c];
    float w = _w[_c];
    const fp16 *__restrict__ const k = _k + _offset;
    const fp16 *__restrict__ const v = _v + _offset;
    fp16 *__restrict__ const y = _y + _offset;

    float aa = _aa[_state_offset];
    float bb = _bb[_state_offset];
    float pp = _pp[_state_offset];
    for (int i = 0; i < T; i++) {
        const int ii = i * C;
        const float kk = float(k[ii]);
        const float vv = float(v[ii]);
        float ww = u + kk;
        float p = max(pp, ww);
        float e1 = exp(pp - p);
        float e2 = exp(ww - p);
        y[ii] = fp16((e1 * aa + e2 * vv) / (e1 * bb + e2));
        ww = w + pp;
        p = max(ww, kk);
        e1 = exp(ww - p);
        e2 = exp(kk - p);
        aa = e1 * aa + e2 * vv;
        bb = e1 * bb + e2;
        pp = p;
    }
    _aa[_state_offset] = aa;
    _bb[_state_offset] = bb;
    _pp[_state_offset] = pp;
}

void cuda_wkv_forward(int B, int T, int C, float *w, float *u, fp16 *k, fp16 *v, fp16 *y, float *aa, float *bb, float *pp) {
    dim3 threadsPerBlock( min(C, 32) );
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_wkv_forward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, y, aa, bb, pp);
}

__half *cast(fp16 *ptr) {
    return reinterpret_cast<__half *>(ptr);
}

__global__ void kernel_mm8_seq(
    const int B, const int N, const int M,
    const __half *__restrict__ const x, const int x_stride,
    const uint8_t *__restrict__ const w, const int w_stride,
    const __half *__restrict__ const mx,
    const __half *__restrict__ const rx,
    const __half *__restrict__ const my,
    const __half *__restrict__ const ry,
    __half *__restrict__ const y, const int y_stride) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int k = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < B && k < M) {
        float y_local = 0;
        for (int j = 0; j < N; ++j) {
            y_local += __half2float(x[i * x_stride + j]) * (
                (float(w[j * w_stride + k]) + 0.5f)
                * __half2float(rx[k]) * __half2float(ry[j])
                + __half2float(mx[k]) + __half2float(my[j])
            );
        }
        y[i * y_stride + k] = __float2half(y_local);
    }
}
void cuda_mm8_seq(int B, int N, int M,
                  fp16 *x, int x_stride,
                  uint8_t *w, int w_stride,
                  fp16 *mx, fp16 *rx,
                  fp16 *my, fp16 *ry,
                  fp16 *y, int y_stride) {
    dim3 blockSize(1, 128);
    dim3 gridSize((B + blockSize.x - 1) / blockSize.x, (M + blockSize.y - 1) / blockSize.y);
    kernel_mm8_seq<<<gridSize, blockSize>>>(
        B, N, M, cast(x), x_stride, w, w_stride,
        cast(mx), cast(rx), cast(my), cast(ry), cast(y), y_stride);
}

#define MM8_ONE_JSPLIT 24
#define MM8_ONE_TILE 1024

__global__ void kernel_mm8_one(
    const int N, const int M,
    const __half *__restrict__ const x,
    const uint8_t *__restrict__ const w, const int w_stride,
    const __half *__restrict__ const mx,
    const __half *__restrict__ const rx,
    const __half *__restrict__ const my,
    const __half *__restrict__ const ry,
    float *__restrict__ const y) {

    const int k = blockIdx.y * blockDim.y + threadIdx.y;
    const int j0 = min(N, blockIdx.x * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));
    const int j1 = min(N, (blockIdx.x + 1) * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));

    if (k < M) {
        float y_local = 0;
        for (int j = j0; j < j1; ++j) {
            y_local += __half2float(x[j]) * (
                (float(w[j * w_stride + k]) + 0.5f)
                * __half2float(rx[k]) * __half2float(ry[j])
                + __half2float(mx[k]) + __half2float(my[j])
            );
        }
        atomicAdd(&y[k], y_local);
    }
}
void cuda_mm8_one(int N, int M,
                  fp16 *x,
                  uint8_t *w, int w_stride,
                  fp16 *mx, fp16 *rx,
                  fp16 *my, fp16 *ry,
                  float *y) {
    dim3 blockSize(1, MM8_ONE_TILE);
    dim3 gridSize(MM8_ONE_JSPLIT, (M + blockSize.y - 1) / blockSize.y);
    kernel_mm8_one<<<gridSize, blockSize>>>(
        N, M, cast(x), w, w_stride,
        cast(mx), cast(rx), cast(my), cast(ry), y);
}
